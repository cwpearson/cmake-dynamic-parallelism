#include <iostream>
#include <cassert>

#include "dp.hu"

#define CUDA_RUNTIME(stmt) checkCuda(stmt, __FILE__, __LINE__);

void checkCuda(hipError_t result, const char *file, const int line) {
    if (result != hipSuccess) {
      std::cerr << file << "@" << line << ": CUDA Runtime Error: " << hipGetErrorString(result) << "\n";
      exit(-1);
    }
  }

int main(void) {

    float *a = nullptr;
    float *b = nullptr;
    float *s = nullptr;

    const size_t n = 100000;

    CUDA_RUNTIME(hipMallocManaged(&a, n * sizeof(*a)));
    CUDA_RUNTIME(hipMallocManaged(&b, n * sizeof(*b)));
    CUDA_RUNTIME(hipMallocManaged(&s, n * sizeof(*s)));

    for (size_t i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = 2*i;
    }

    launcher<<<1,1>>>(s,a,b,n);
    CUDA_RUNTIME(hipGetLastError());
    CUDA_RUNTIME(hipDeviceSynchronize());

    for (size_t i = 0; i < n; ++i) {
        assert(s[i] == a[i] + b[i]);
    }

    CUDA_RUNTIME(hipFree(s));
    CUDA_RUNTIME(hipFree(b));
    CUDA_RUNTIME(hipFree(a));

    return 0;
}